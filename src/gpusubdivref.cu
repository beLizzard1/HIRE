#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <inttypes.h>
#include <math.h>

#include <tiffio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

__global__ void reducedhologramkernel(hipComplex *gpureducedhologram, hipComplex *gpureferencewave, float *gpusubtractedimage, unsigned int width){

	int bidx = blockIdx.x;
	int bidy = blockIdx.y;
	int offset;
	offset = (bidy * width) + bidx;

	gpureducedhologram[offset].x = gpusubtractedimage[offset] * ( 1 * gpureferencewave[offset].x);
	gpureducedhologram[offset].y = gpusubtractedimage[offset] * (-1 * gpureferencewave[offset].y);

}


extern "C" int gpusubdivref(hipComplex *reducedhologram, float *subtractedimage, hipComplex *referencewave,unsigned int width, unsigned int height){

	        dim3 threadsPerBlock(1,1);
	        dim3 numBlock(width/threadsPerBlock.x, height/threadsPerBlock.y);

		hipComplex *gpureducedhologram, *gpureferencewave;
		float *gpusubtractedimage;

		hipMalloc(&gpureducedhologram, sizeof(hipComplex) * width * height);
		hipMalloc(&gpureferencewave, sizeof(hipComplex) * width * height);
		hipMalloc(&gpusubtractedimage, sizeof(float) * width * height);

		hipMemcpy(gpureferencewave, referencewave, sizeof(hipComplex) * width * height, hipMemcpyHostToDevice);
		hipMemcpy(gpusubtractedimage, subtractedimage, sizeof(float) * width * height, hipMemcpyHostToDevice);

		hipDeviceSynchronize();
		printf("ReducedHologram errors allocating and copying memory errors (?): %s\n", hipGetErrorString(hipGetLastError()));

	reducedhologramkernel<<<numBlock, threadsPerBlock>>>(gpureducedhologram, gpureferencewave, gpusubtractedimage, width);

		hipDeviceSynchronize();
		printf("Running the kernel errors (?): %s\n", hipGetErrorString(hipGetLastError()));

		hipMemcpy(reducedhologram, gpureducedhologram, sizeof(hipComplex) * width * height, hipMemcpyDeviceToHost);	
		hipDeviceSynchronize();
		printf("Copying result from GPU, errors (?): %s\n", hipGetErrorString(hipGetLastError()));

		hipFree(gpureducedhologram);
		hipFree(gpureferencewave);
		hipFree(gpusubtractedimage);

		hipDeviceReset();
	

	return(0);

}
