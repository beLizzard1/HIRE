#include <stdio.h>
#include <stdlib.h>

#include <inttypes.h>
#include <math.h>

#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>

extern "C" int gpufouriertransform(hipComplex *original, hipComplex *transform, unsigned int width, unsigned int height){

	hipfftHandle plan;
	hipfftPlan2d(&plan,width, height, HIPFFT_C2C);

	hipComplex *gpuoriginal, *gputransform;

	printf("Starting the gpu FFT\n");

	hipMalloc(&gpuoriginal, sizeof(hipComplex) * width * height);
	hipMalloc(&gputransform, sizeof(hipComplex) * width * height);

	hipDeviceSynchronize();
        printf("Allocating Memory errors (?): %s\n", hipGetErrorString(hipGetLastError()));

	hipMemcpy(gpuoriginal, original, sizeof(hipComplex) * width * height, hipMemcpyHostToDevice);

	hipDeviceSynchronize();
        printf("Copying Memory errors (?): %s\n", hipGetErrorString(hipGetLastError()));
	
	hipfftExecC2C(plan,gpuoriginal,gputransform, -1);

	hipMemcpy(transform, gputransform, sizeof(hipComplex) * width * height, hipMemcpyDeviceToHost);		
	hipDeviceSynchronize();
        printf("Copying back, Memory errors (?): %s\n", hipGetErrorString(hipGetLastError()));

/*
	for(offset = 0; offset < (width * height); offset ++){
		printf("%g%+g\n", transform[offset].x, transform[offset].y);
	}

*/

	return(0);
}

extern "C" int floatfft(float* original,hipComplex* transform,unsigned int width, unsigned int height){


	hipfftHandle plan;
	hipfftPlan2d(&plan,width, height, HIPFFT_R2C);

	hipfftReal *gpuoriginal;
	hipComplex  *gputransform;


	hipMalloc(&gpuoriginal, sizeof(hipComplex) * width * height);
	hipMalloc(&gputransform, sizeof(hipfftReal) * width * height);

	hipDeviceSynchronize();

	hipMemcpy(gpuoriginal, original, sizeof(hipfftReal) * width * height, hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	hipfftExecR2C(plan, gpuoriginal, gputransform);

	hipMemcpy(transform, gputransform, sizeof(hipComplex) * width * height, hipMemcpyDeviceToHost);


	return(0);

}
