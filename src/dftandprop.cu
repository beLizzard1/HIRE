#include <complex.h>
#include <hip/hip_complex.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#define PI 3.1415926535

__global__ void kspacedistkernel(float* kspacedist, unsigned int width, unsigned int height){

	int bidx = blockIdx.x;
	int bidy = blockIdx.y;
	int offset;

	offset = (bidx * width) + bidy;

	float xkspace, ykspace;

	xkspace = ((2 * PI ) / 6.45 ) / (width);
	ykspace = ((2 * PI ) / 6.45 ) / (height);

	kspacedist[offset] = sqrtf( powf(xkspace*(bidx - 696),2) + powf(ykspace*(bidy - 520),2));

}

extern "C" int gpudftwithprop(hipComplex *reducedhologram, unsigned int width, unsigned int height, float startz, float endz){

	printf("Starting Discrete Fourier Transform and Propagation Stuff\n");
	dim3 threadsPerBlock(1,1);
        dim3 numBlock(width/threadsPerBlock.x, height/threadsPerBlock.y);
	hipComplex *gpureducedhologram, *gputransformed, *transformed;		
	float *kspacedist, *temp;
	int offset;
/*
	for(offset = 0; offset < width * height; offset++){
		printf("Reduced Hologram: %f%+fi\n",reducedhologram[offset].x, reducedhologram[offset].y);
	}
*/
	transformed = (hipfftComplex *)malloc(sizeof(hipComplex)*width*height);

	hipMalloc(&gpureducedhologram, sizeof(hipComplex) * width * height);
	hipMalloc(&gputransformed, sizeof(hipComplex) * width * height);
	hipMemcpy(gpureducedhologram, reducedhologram, sizeof(hipComplex) * width * height, hipMemcpyHostToDevice);
	
	printf("Allocating memory on the GPU so that the cuFFT library can access it during the operation\n");
	
	hipfftHandle plan;
	hipfftPlan2d(&plan, width, height, HIPFFT_C2C);

	hipfftExecC2C(plan,gpureducedhologram,gputransformed, -1);

	hipMemcpy(transformed, gputransformed, sizeof(hipComplex) * width * height, hipMemcpyDeviceToHost);
/*
	for(offset = 0; offset < (width * height); offset++){
		printf("%f %+fi\n", transformed[offset].x, transformed[offset].y);
	} 
*/
	hipFree(gpureducedhologram);
	hipFree(gputransformed);	
	
	hipMalloc(&kspacedist, sizeof(float) * width * height);
	temp = (float *)malloc(sizeof(float) * width * height);

	kspacedistkernel<<<numBlock, threadsPerBlock>>>(kspacedist,width, height);

	hipMemcpy(temp, kspacedist, sizeof(float) * width * height, hipMemcpyDeviceToHost);

/*	for(offset = 0; offset < (width * height); offset ++){
		printf("k-space dist: %f\n", temp[offset]);
	
	}*/

	hipfftDestroy(plan);

	return(0);
}
