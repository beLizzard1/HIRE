#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <inttypes.h>
#include <math.h>
#include <tiffio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

__global__ void gpurefwavecalc(hipComplex *gpureferencewave, float *gpudistancegrid, float *gpuimage2f, float k, float width){

	int bidx = blockIdx.x;
	int bidy = blockIdx.y;
	int offset;
	float sincomp, coscomp;

	offset = (bidy * width) + bidx;

	sincosf(k * gpudistancegrid[offset], &sincomp, &coscomp);

	gpureferencewave[offset].x = (gpuimage2f[offset] * coscomp);
	gpureferencewave[offset].y = (gpuimage2f[offset] * sincomp);


}

extern "C" int gpurefwavecalc(hipComplex *refwave, float *image2f, float *distancegrid, float k, unsigned int width, unsigned int height){

	dim3 threadsPerBlock(1,1);
	dim3 numBlock(width/threadsPerBlock.x, height/threadsPerBlock.y);

	hipComplex *gpureferencewave;
	float *gpudistancegrid, *gpuimage2f;

	hipMalloc(&gpureferencewave, sizeof(hipComplex) * width * height);
	hipMalloc(&gpudistancegrid, sizeof(float) * width * height);
	hipMalloc(&gpuimage2f, sizeof(float) * width * height);

	hipMemcpy(gpudistancegrid, distancegrid, sizeof(float) * width * height, hipMemcpyHostToDevice);
	hipMemcpy(gpuimage2f, image2f, sizeof(float) * width * height, hipMemcpyHostToDevice);

	hipDeviceSynchronize();
	printf("Allocating Memory errors (?): %s\n", hipGetErrorString(hipGetLastError()));

	gpurefwavecalc<<<numBlock, threadsPerBlock>>>(gpureferencewave, gpudistancegrid, gpuimage2f, k, width);
	hipDeviceSynchronize();
	printf("ReferenceWaveCalcKernel errors(?): %s\n", hipGetErrorString(hipGetLastError()));

	hipMemcpy(refwave, gpureferencewave, sizeof(hipComplex) * width * height, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	printf("ReferenceWaveCalcKernel errors(?): %s\n", hipGetErrorString(hipGetLastError()));

	hipFree(gpureferencewave);
	hipFree(gpudistancegrid);
	hipFree(gpuimage2f);
	hipDeviceReset();
	return(0);
}
