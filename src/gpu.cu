#include <complex.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>


__global__ void sqrtkernel(float* gpudata, float* gpuresult, unsigned int width){

	int tidx = blockIdx.x;
	int tidy = blockIdx.y;
	int offset;

	offset = (tidy * width)+ tidx;
	gpuresult[offset] = sqrtf(gpudata[offset]);

}

__global__ void distancekernel(float* gpudistance, unsigned int width, unsigned int height, float realz){
	int xcoord = blockIdx.x;
	int ycoord = blockIdx.y;
	int offset;
	float xcontrib, ycontrib, zcontrib, pixelsize;
	pixelsize = 6.45;

	offset = (ycoord * width) + xcoord;
	xcontrib = ((float)xcoord - (((float)width / 2)-1)) * pixelsize;
	ycontrib = ((float)ycoord - (((float)height / 2)-1)) * pixelsize;
	zcontrib = realz;

	gpudistance[offset] = sqrtf( (xcontrib * xcontrib) + (ycontrib * ycontrib) + (zcontrib * zcontrib));
		
}

extern "C" int gpudistance(float *distance, unsigned int width, unsigned int height){
	dim3 threadsPerBlock(1,1);
	dim3 numBlock(width/threadsPerBlock.x, height/threadsPerBlock.y);
	float realz;
	float *gpudistance;

	realz = 43048; /* Distance from pinhold in Z axis */

	hipMalloc(&gpudistance, sizeof(hipComplex) * (width * height));
	distancekernel<<<numBlock, threadsPerBlock >>>(gpudistance, width, height, realz);	
	hipDeviceSynchronize();
	printf("GPU Distance: %s\n", hipGetErrorString(hipGetLastError()));
	
	hipMemcpy(distance, gpudistance, sizeof(float) * width * height, hipMemcpyDeviceToHost);
/*
	int offset;
	for (offset = 0; offset < (width * height); offset ++){
		printf("GPU Distance Function: %f \n", distance[offset]);
	}
*/	
	hipFree(gpudistance);
	hipDeviceReset();

	return(0);

}

extern "C" int gpusqrt(float *data, unsigned int width, unsigned int height){
	dim3 threadsPerBlock(1,1);
	dim3 numBlock(width/threadsPerBlock.x, height/threadsPerBlock.y);
	float *gpudata, *gpuresult;	
	int offset;

	hipMalloc(&gpudata, sizeof(float) * (width * height));
	hipMalloc(&gpuresult, sizeof(float) * (width * height));	
	hipMemcpy(gpudata, data, sizeof(float) * (width * height), hipMemcpyHostToDevice);

	sqrtkernel<<<numBlock, threadsPerBlock >>>(gpudata,gpuresult, width);
	hipDeviceSynchronize();
	printf("GPUSQRT: %s\n", hipGetErrorString(hipGetLastError()));

	hipFree(gpudata);

	hipMemcpy(data, gpuresult, sizeof(float) * (width * height), hipMemcpyDeviceToHost);

/*	for(offset = 0; offset < (width * height); offset++){
		printf("GPU Sqrt Function Results: %f\n", data[offset]);

	} */

 	/*	Works Fine upto here */

	hipFree(gpuresult);
	hipDeviceReset();

	return(0);
}

__global__ void wavecalckernel(hipComplex *gpureferencewave, float *gpudistancegrid, float *gpudata, float k, unsigned int width){

        int bidx = blockIdx.x;
        int bidy = blockIdx.y;
        int offset;
        float sincomp, coscomp;

        offset = (bidy * width) + bidx;

        sincosf(k * gpudistancegrid[offset], &sincomp, &coscomp);

        gpureferencewave[offset].x = (gpudata[offset] * coscomp)/(gpudistancegrid[offset]);
        gpureferencewave[offset].y = (gpudata[offset] * sincomp)/(gpudistancegrid[offset]);

}

extern "C" int gpurefwavecalc(hipComplex *referencewave,float *data,float *distancegrid,float k,unsigned int width,unsigned int height){

	dim3 threadsPerBlock(1,1);
	dim3 numBlock(width/threadsPerBlock.x, height/threadsPerBlock.y);
	printf("K Value is: %g\n", k);	

	hipComplex *gpureferencewave;
	float *gpudistancegrid, *gpudata;

	printf("Allocating the memory on the GPU\n");
	
	hipMalloc(&gpureferencewave, sizeof(hipComplex) * (width * height));
	hipMalloc(&gpudistancegrid, sizeof(float) * (width * height));
	hipMalloc(&gpudata, sizeof(float) * width * height);
	
	printf("Copying the CPU memory onto the newly allocated GPU\n");

	hipMemcpy(gpudistancegrid, distancegrid, sizeof(float) * width * height,hipMemcpyHostToDevice);
	hipMemcpy(gpudata, data, sizeof(float) * width * height,hipMemcpyHostToDevice);

	printf("Starting a GPU Kernel\n");

	wavecalckernel<<<numBlock, threadsPerBlock>>>(gpureferencewave, gpudistancegrid, gpudata, k, width);
	hipDeviceSynchronize();

	printf("ReferenceWaveCalc: %s\n", hipGetErrorString(hipGetLastError()));

/*	cudaMemcpy(referencewave, gpureferencewave, sizeof(cuComplex) * width * height, cudaMemcpyDeviceToHost); */

/*	int offset;
	for(offset = 0; offset < (width * height); offset++){
		printf("Reference Wave: %f%+f\n", referencewave[offset].x, referencewave[offset].y);
	} */

	hipDeviceReset();

	return(0);
};

__global__ void reducedhologramkernel(hipComplex *gpureducedhologram, hipComplex *gpurefwave, float* gpusubimage, unsigned int width){

	int bidx = blockIdx.x;
	int bidy = blockIdx.y;
	int offset;
	offset = (bidy * width) + bidx;

/*	gpureducedhologram[offset].x = gpusubimage[offset] / (1 * gpurefwave[offset].x);
	gpureducedhologram[offset].y = gpusubimage[offset] / (-1 * gpurefwave[offset].y);
*/

	gpureducedhologram[offset].x = 1;
}

extern "C" int subimagedivref(hipComplex *reducedhologram, float *subimage, hipComplex *refwave, unsigned int width, unsigned int height){
	int offset;
        dim3 threadsPerBlock(1,1);
        dim3 numBlock(width/threadsPerBlock.x, height/threadsPerBlock.y);

	printf("Starting the division of the subtracted image and the reference wave\n");

/*	for(offset = 0; offset < (width * height); offset ++){
		printf("SubImage: %f\n",subimage[offset]);
		printf("refwave: %f%+fi\n", refwave[offset].x, refwave[offset].y);
	} */


	hipComplex *gpureducedhologram, *gpurefwave;
	float *gpusubimage;
	printf("Starting to allocate the memory\n");
	
	hipMalloc(&gpureducedhologram, sizeof(hipComplex)*(width*height));
	hipMalloc(&gpurefwave, sizeof(hipComplex)*(width*height));
	hipMalloc(&gpusubimage, sizeof(float)*(width*height));
	
	printf("Finished Allocating the Memory\n");	


	hipMemcpy(gpurefwave,refwave, sizeof(hipComplex)*(width*height), hipMemcpyHostToDevice);
	hipMemcpy(gpusubimage, subimage, sizeof(float)*(width * height), hipMemcpyHostToDevice);

	reducedhologramkernel<<<numBlock, threadsPerBlock>>>(gpureducedhologram,gpurefwave,gpusubimage, width);
	hipDeviceSynchronize();
	
	printf("ReducedHologram: %s\n", hipGetErrorString(hipGetLastError()));
	
	hipMemcpy(reducedhologram, gpureducedhologram, sizeof(hipComplex) * width * height, hipMemcpyDeviceToHost);

/*	for(offset = 0; offset < (width * height); offset ++){
		printf("%f%+fi \n", reducedhologram[offset].x, reducedhologram[offset].y);
	} */

	hipFree(gpureducedhologram);
	hipFree(gpurefwave);
	hipFree(gpusubimage);

	hipDeviceReset();

	return(0);

}
