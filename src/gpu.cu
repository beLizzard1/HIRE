#include <complex.h>
#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void sqrtkernel(float* gpudata, float* gpuresult, unsigned int width){

	int tidx = blockIdx.x;
	int tidy = blockIdx.y;
	int offset;

	offset = (tidy * width)+ tidx;
	gpuresult[offset] = sqrtf(gpudata[offset]);

}

__global__ void distancekernel(float* gpudistance, unsigned int width, unsigned int height, float realz){
	int xcoord = blockIdx.x;
	int ycoord = blockIdx.y;
	int offset;
	
	offset = (ycoord * width) + xcoord;
		
	gpudistance[offset] = sqrtf( pow(((xcoord - 696)*6.45),2) + pow(((ycoord - 520)*6.45),2) + pow(realz,2) );

}

__global__ void wavecalckernel(hipComplex** refwave, float* data, float* distance, float k, unsigned int width){

	int xcoord = blockIdx.x;
	int ycoord = blockIdx.y;
	int offset;
	float sinvalue, cosvalue;

	offset = (ycoord * width) + xcoord;
	sincosf(k * distance[offset], &sinvalue, &cosvalue);
	
	refwave[offset]->x = data[offset] * cosvalue;
	refwave[offset]->y = data[offset] * sinvalue;
}

extern "C" int referencephase(float *data, unsigned int width, unsigned int height){
	dim3 threadsPerBlock(1,1);
	dim3 numBlock(width/threadsPerBlock.x, height/threadsPerBlock.y);
	float realz, k;
	float *distance, *gpudistance, *gpudata;
	int offset;

	k = 2 * M_PI / 0.780241;

	realz = 43048; /* Distance from pinhold in Z axis */

	distance = (float*)malloc(width * height * sizeof(float));
	hipMalloc(&gpudistance, sizeof(hipComplex) * (width * height));
	distancekernel<<<numBlock, threadsPerBlock >>>(gpudistance, width, height, realz);	
	hipMemcpy(distance, gpudistance, sizeof(float) * width * height, hipMemcpyDeviceToHost);

	hipComplex *gpurefwave, *refwave;
	hipMalloc(&gpurefwave, sizeof(hipComplex) * (width * height));
	refwave = (hipComplex *)calloc(width * height, sizeof(hipComplex));
	hipMalloc(&gpudata, sizeof(float) * width * height);
	hipMemcpy(gpudata,data, sizeof(float) * (width * height),hipMemcpyHostToDevice);

	wavecalckernel<<<numBlock, threadsPerBlock >>>(&gpurefwave, gpudata, gpudistance, k, width);
	hipMemcpy(gpurefwave,refwave,sizeof(hipComplex) * width * height, hipMemcpyDeviceToHost);
	
	for(offset = 0; offset < (width * height); offset++){
		printf("%g + %gi\n", refwave[offset].x, refwave[offset].y);
	}

	hipFree(gpurefwave);
	return(0);

}

extern "C" int gpusqrt(float *data, unsigned int width, unsigned int height){
	dim3 threadsPerBlock(1,1);
	dim3 numBlock(width/threadsPerBlock.x, height/threadsPerBlock.y);
	float *gpudata, *gpuresult;	

	hipMalloc(&gpudata, sizeof(float) * (width * height));
	hipMalloc(&gpuresult, sizeof(float) * (width * height));	
	hipMemcpy(gpudata, data, sizeof(float) * (width * height), hipMemcpyHostToDevice);

	sqrtkernel<<<numBlock, threadsPerBlock >>>(gpudata,gpuresult, width);
	hipFree(gpudata);

	hipMemcpy(data, gpuresult, sizeof(float) * (width * height), hipMemcpyDeviceToHost);
	
	hipFree(gpuresult);
	return(0);
}
