#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include "complexnumbers.h"

__global__ void sqrtkernel(float* gpudata, float* gpuresult, unsigned int width){

	int tidx = blockIdx.x;
	int tidy = blockIdx.y;
	int offset;

	offset = (tidy * width)+ tidx;
	gpuresult[offset] = sqrtf(gpudata[offset]);

}

__global__ void distancekernel(float* gpudistance, unsigned int width, unsigned int height, float realz){
	int xcoord = blockIdx.x;
	int ycoord = blockIdx.y;
	int offset;
	
	offset = (ycoord * width) + xcoord;
		
	gpudistance[offset] = sqrtf( pow(((xcoord - 696)*6.45),2) + pow(((ycoord - 520)*6.45),2) + pow(realz,2) );

}

extern "C" int referencephase(float *data, unsigned int width, unsigned int height){
	dim3 threadsPerBlock(1,1);
	dim3 numBlock(width/threadsPerBlock.x, height/threadsPerBlock.y);
	unsigned int x, y, offset;
	float realz, k;
	float *distance, *gpudistance;
	
	k = 2 * M_PI / 0.780241;

	realz = 43048; /* Distance from pinhold in Z axis */

	distance = (float*)malloc(width * height * sizeof(float));
	hipMalloc(&gpudistance, sizeof(float) * (width * height));
	distancekernel<<<numBlock, threadsPerBlock >>>(gpudistance, width, height, realz);	
	hipMemcpy(distance, gpudistance, sizeof(float) * width * height, hipMemcpyDeviceToHost);
	hipFree(gpudistance);
	
	Complex ReferenceWave;

	referencewave<<<numBlock, threadsPerBlock>>>(ReferenceWave, k, distance, data, width, height);			

	return(0);

}

extern "C" int gpusqrt(float *data, unsigned int width, unsigned int height){
	dim3 threadsPerBlock(1,1);
	dim3 numBlock(width/threadsPerBlock.x, height/threadsPerBlock.y);
	float *gpudata, *gpuresult;	

	hipMalloc(&gpudata, sizeof(float) * (width * height));
	hipMalloc(&gpuresult, sizeof(float) * (width * height));	
	hipMemcpy(gpudata, data, sizeof(float) * (width * height), hipMemcpyHostToDevice);

	sqrtkernel<<<numBlock, threadsPerBlock >>>(gpudata,gpuresult, width);
	hipFree(gpudata);

	hipMemcpy(data, gpuresult, sizeof(float) * (width * height), hipMemcpyDeviceToHost);
	
	hipFree(gpuresult);
	return(0);
}
